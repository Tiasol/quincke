#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "QuinckeForceComputeGPU.cuh"
#include "hoomd/RandomNumbers.h"
#include "hoomd/RNGIdentifiers.h"
using namespace hoomd;

#include <assert.h>

/*! \file QuinckeForceCompute.cu
    \brief CUDA kernels for QuinckeForceCompute
*/

// First, the kernel code for zeroing the velocities on the GPU
//! Kernel that zeroes velocities on the GPU
/*! \param d_vel Velocity-mass array from the ParticleData
    \param N Number of particles

    This kernel executes one thread per particle and zeros the velocity of each. It can be run with any 1D block size
    as long as block_size * num_blocks is >= the number of particles.
*/

__global__ void gpu_compute_active_force_set_forces_kernel(const unsigned int group_size,
                                                    unsigned int *d_rtag,
                                                    unsigned int *d_groupTags,
                                                    Scalar4 *d_force,
                                                    Scalar4 *d_torque,
                                                    Scalar4 *d_orientation,
                                                    Scalar Dpassive,
                                                    Scalar Dactive,
                                                    Scalar Ee,
                                                    Scalar Ecut,
                                                    Scalar rcut,
                                                    Scalar sigma21,
                                                    Scalar H,
                                                    Scalar epsilon,
                                                    const unsigned int N)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= group_size)
        return;

    unsigned int tag = d_groupTags[group_idx];
    unsigned int idx = d_rtag[tag];


    // for now do nothing
    //quat<Scalar> quati(h_orientation.data[idx]);
    //Scalar3 f = make_scalar3(0,0,m_params);
    //vec3<Scalar> fi = rotate(quati, vec3<Scalar>(f));
    //d_force[idx].x = fi.x;
    //d_force[idx].y = fi.y;
    //d_force[idx].z = fi.z;  

    
    }



/*! \param d_vel Velocity-mass array from the ParticleData
    \param N Number of particles
    This is just a driver for gpu_zero_velocities_kernel(), see it for the details
*/
hipError_t gpu_compute_quincke_force_set_forces(const unsigned int group_size,
                                                    unsigned int *d_rtag,
                                                    unsigned int *d_groupTags,
                                                    Scalar4 *d_force,
                                                    Scalar4 *d_torque,
                                                    Scalar4 *d_orientation,
                                                    Scalar Dpassive,
                                                    Scalar Dactive,
                                                    Scalar Ee,
                                                    Scalar Ecut,
                                                    Scalar rcut,
                                                    Scalar sigma21,
                                                    Scalar H,
                                                    Scalar epsilon,
                                                    const unsigned int N,
                                                    unsigned int block_size)
    {
    // setup the grid to run the kernel
    dim3 grid( group_size / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    hipMemset(d_force, 0, sizeof(Scalar4)*N);
    gpu_compute_quincke_force_set_forces_kernel<<< grid, threads>>>( group_size,
                                                                    d_rtag,
                                                                    d_groupTags,
                                                                    d_force,
                                                                    d_torque,
                                                                    d_orientation,
                                                                    Dpassive,
                                                                    Dactive,
                                                                    Ee,
                                                                    Ecut,
                                                                    rcut,
                                                                    sigma21,
                                                                    H,
                                                                    epsilon,
                                                                    N);
    return hipSuccess;
    }